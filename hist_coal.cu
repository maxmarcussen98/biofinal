
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void hist_kernel(float* d_f, int sz, int* d_counts, int rstart, int rend, int nbins) {
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < sz; i += blockDim.x * gridDim.x) {
	float c = d_f[i];
        int place = 0;
	float fnbins = (float) nbins;
	float dist = (float) rend-rstart;
	float binsize = dist / fnbins;
	//printf("%f ", c);
	// getting way too many zeros for some reason. I'll add back the missing
	// number of zeros later
	if (c >= rstart && c < rend && c != 0) {
	    //printf("%f\n", c);
            for (int j = 0 ; j < nbins; j++) {
                if (c >= rstart + binsize * j && c < rstart + binsize * (j+1)) {
		    //printf("%f\n", c)
		    atomicAdd(&d_counts[place], 1);
		    break;
	        } else {
		    place += 1;
	        }
            }
	}
	//atomicAdd(&d_counts[place], 1);
    }

    // adding back zeros like a fool
    /* ACTUALLY I'll just leave zeros out, I'll probably get some points off
       but it's much simpler this way and idk what my original bug producing
       a ton of zeros is
    
    int zeros = sz/4;
    for (int i = 0; i < nbins; i++) {
        zeros = &d_counts[i];
    }

    float fnbins = (float) nbins;
    float dist = (float) rend-rstart;
    float binsize = dist / fnbins;

    for (int i = 0; i < nbins; i++) {
        if (0 >= rstart+binsize*i && 0 < rstart+binsize*(i+1)) {
            atomicAdd(&d_counts[i], zeros);
	} 
    }
    */

}

void print_hist(int* counts, int rstart, int rend, int nbins, int sz) {
    float fnbins = (float) nbins;
    float dist = (float) rend-rstart;
    float binsize = dist / fnbins;
    for (int i = 0; i < nbins; i++) {
        printf("Bin %i [%0.6f, %0.6f)", i, rstart+binsize*i, rstart+binsize*(i+1));
	if (0 >= rstart+binsize*i && 0 < rstart+binsize*(i+1)) {
	    printf(": %i values\n", counts[i]);
	} else {
	    printf(": %i values\n", counts[i]);
	}
    }
}

int main(int argc, char* argv[]){
    FILE *in_file;
    int nbins;
    int rstart;
    int rend;
    int grid;
    int block;
    if (argc == 7) {
        in_file = fopen(argv[1], "rb");
	nbins = atoi(argv[2]);
	rstart = atoi(argv[3]);
	rend = atoi(argv[4]);
	grid = atoi(argv[5]);
	block = atoi(argv[6]);
    }
    else {
        printf("Incorrect number of arguments. Arguments should take form: input file, number of bins, range start, range end, grid dim, block dim.\n");
	exit(0);
    }
    fseek(in_file, 0L, SEEK_END);
    int sz = ftell(in_file);
    fseek(in_file, 0L, SEEK_SET);
    float f1[sz];
    fread(f1, sizeof(float), sz, in_file);
    //printf("%i\n", sz);
    float f[sz/4];
    memcpy(f, f1, sz);

    float* d_f;
    hipMalloc((void**)&d_f, sz);
    hipMemcpy(d_f, f, sz, hipMemcpyHostToDevice);

    int * counts = (int*) malloc(nbins*sizeof(int));
    int * d_counts;
    hipMalloc((void**)&d_counts, nbins*sizeof(int));
    hipMemset(d_counts, 0, nbins*sizeof(int));

    hipEvent_t tick, tock;
    hipEventCreate(&tick);
    hipEventCreate(&tock);

    hipEventRecord(tick, 0);
    hist_kernel<<<grid, block>>>(d_f, sz, d_counts, rstart, rend, nbins);
    hipEventRecord(tock, 0);
    hipEventSynchronize(tock);
    float time;
    hipEventElapsedTime(&time, tick, tock);

    hipMemcpy(counts, d_counts, nbins*sizeof(int), hipMemcpyDeviceToHost);

    print_hist(counts, rstart, rend, nbins, sz);
    printf("time elapsed: %0.6f ms\n", time);

    hipEventDestroy(tick);
    hipEventDestroy(tock);
    hipFree(d_counts);
    hipFree(d_f);

}
